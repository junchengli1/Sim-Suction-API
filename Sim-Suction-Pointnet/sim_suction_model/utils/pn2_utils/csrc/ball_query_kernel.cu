#include "hip/hip_runtime.h"
/* CUDA Implementation for ball query*/
#ifndef _BALL_QUERY_KERNEL
#define _BALL_QUERY_KERNEL

#include <cmath>
#include <vector>

#include <ATen/ATen.h>
//#include <THC/THC.h>

// NOTE: AT_ASSERT has become AT_CHEAK on master after 0.4.
// NOTE: AT_CHEAK has become TORCH_CHECK on master after 1.x.
#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)
#define CHECK_EQ(x, y) TORCH_CHECK(x == y, #x " does not equal to " #y)
// #define CHECK_GT(x, y) TORCH_CHECK(x > y, #x " is not greater than " #y)
#define AT_CUDA_CHECK TORCH_CHECK
#define MAX_THREADS uint64_t(512)

inline uint64_t get_block(int64_t x) {
  int cnt = 0;
  x -= 1;
  while (x > 0) {
    x = x >> 1;
    cnt += 1;
  }
  return std::min(uint64_t(1) << cnt, MAX_THREADS);
}

template <typename scalar_t, typename index_t>
__global__ void BallQueryKernel(
    index_t* __restrict__ index,
    index_t* __restrict__ count,
    const scalar_t *__restrict__ points,
    const int64_t num_points,
    const scalar_t *__restrict__ centroids,
    const int64_t num_centroids,
    const scalar_t radius,
    const int64_t num_neighbours) {
  const int batch_index = blockIdx.x;
  index += batch_index * num_centroids * num_neighbours;
  count += batch_index * num_centroids;
  points += batch_index * num_points * 3;
  centroids += batch_index * num_centroids * 3;
  
  scalar_t radius_square = radius * radius;
  for (int i = threadIdx.x; i < num_centroids; i += blockDim.x) {
    int offset1 = i * 3;
    int offset3 = i * num_neighbours;
    scalar_t x1 = centroids[offset1 + 0];
    scalar_t y1 = centroids[offset1 + 1];
    scalar_t z1 = centroids[offset1 + 2];
    index_t cnt = 0;
    for (int j = 0; j < num_points && cnt < num_neighbours; ++j) {
      int offset2 = j * 3;
      scalar_t x2 = points[offset2 + 0];
      scalar_t y2 = points[offset2 + 1];
      scalar_t z2 = points[offset2 + 2];
      scalar_t dist = (x2-x1)*(x2-x1)+(y2-y1)*(y2-y1)+(z2-z1)*(z2-z1);
      if (dist < radius_square) {
        if (cnt == 0) {
          for (int k = 0; k < num_neighbours; ++k) {
            index[offset3 + k] = j;
          }
        } else {
          index[offset3 + cnt] = j;
        }
        ++cnt;
      }
    }
    count[i] = cnt;
  }
}

/*
Only forward is required.
Input:
  points: (B, 3, N1)
  centroids: (B, 3, N2)
  raidus: scalar
  num_neighbours: int
Output:
  index: (B, N2, N3)
  count: (B, N2)
*/
std::vector<at::Tensor> BallQuery(
    const at::Tensor points,
    const at::Tensor centroids,
    const float radius,
    const int64_t num_neighbours) {

  const auto batch_size = points.size(0);
  const auto num_points = points.size(2);
  const auto num_centroids = centroids.size(2);

  // Sanity check
  CHECK_CUDA(points);
  CHECK_CUDA(centroids);
  CHECK_EQ(points.size(1), 3);
  CHECK_EQ(centroids.size(1), 3);
  
  auto points_trans = points.transpose(1, 2).contiguous();  // (B, N1, 3)
  auto centroids_trans = centroids.transpose(1, 2).contiguous();  // (B, N2, 3)

  // Allocate new space for output
  auto index = at::zeros({batch_size, num_centroids, num_neighbours}, points.type().toScalarType(at::kLong));
  index.set_requires_grad(false);
  auto count = at::zeros({batch_size, num_centroids}, index.type());
  CHECK_CUDA(index); CHECK_CONTIGUOUS(index);
  CHECK_CUDA(count); CHECK_CONTIGUOUS(count);

  const auto block = get_block(num_centroids);

  AT_DISPATCH_FLOATING_TYPES(points.type(), "BallQuery", ([&] {
    BallQueryKernel<scalar_t, int64_t>
      <<<batch_size, block>>>(
      index.data<int64_t>(),
      count.data<int64_t>(),
      points_trans.data<scalar_t>(),
      num_points,
      centroids_trans.data<scalar_t>(),
      num_centroids,
      (scalar_t)radius,
      num_neighbours);
  }));

  TORCH_CHECK(hipGetLastError() == hipSuccess);

  return std::vector<at::Tensor>({index, count});
}

#endif
